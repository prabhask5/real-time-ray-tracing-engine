
#include <hip/hip_runtime.h>
#ifdef USE_CUDA

#include "../../core/Hittable.cuh"
#include "RotateY.cuh"

CudaRotateY cuda_make_rotate_y(const CudaHittable *object,
                               double angle_degrees) {
  CudaRotateY rotate;
  rotate.object = object;

  double radians = cuda_degrees_to_radians(angle_degrees);
  rotate.sin_theta = sin(radians);
  rotate.cos_theta = cos(radians);
  rotate.bbox = cuda_hittable_get_bounding_box(*object);

  CudaPoint3 min = cuda_make_vec3(CUDA_INF, CUDA_INF, CUDA_INF);
  CudaPoint3 max = cuda_make_vec3(-CUDA_INF, -CUDA_INF, -CUDA_INF);

  for (int i = 0; i < 2; i++) {
    for (int j = 0; j < 2; j++) {
      for (int k = 0; k < 2; k++) {
        double x = i * rotate.bbox.x.max + (1 - i) * rotate.bbox.x.min;
        double y = j * rotate.bbox.y.max + (1 - j) * rotate.bbox.y.min;
        double z = k * rotate.bbox.z.max + (1 - k) * rotate.bbox.z.min;

        double new_x = rotate.cos_theta * x + rotate.sin_theta * z;
        double new_z = -rotate.sin_theta * x + rotate.cos_theta * z;

        CudaVec3 tester = cuda_make_vec3(new_x, y, new_z);
        for (int c = 0; c < 3; c++) {
          cuda_vec3_set(min, c,
                        fmin(cuda_vec3_get(min, c), cuda_vec3_get(tester, c)));
          cuda_vec3_set(max, c,
                        fmax(cuda_vec3_get(max, c), cuda_vec3_get(tester, c)));
        }
      }
    }
  }

  rotate.bbox = cuda_make_aabb(min, max);
  return rotate;
}

__device__ bool cuda_rotate_y_hit(const CudaRotateY &rotate, const CudaRay &ray,
                                  CudaInterval t_values, CudaHitRecord &record,
                                  curandState *rand_state) {
  CudaPoint3 origin = cuda_make_vec3(
      rotate.cos_theta * ray.origin.x - rotate.sin_theta * ray.origin.z,
      ray.origin.y,
      rotate.sin_theta * ray.origin.x + rotate.cos_theta * ray.origin.z);

  CudaVec3 direction = cuda_make_vec3(
      rotate.cos_theta * ray.direction.x - rotate.sin_theta * ray.direction.z,
      ray.direction.y,
      rotate.sin_theta * ray.direction.x + rotate.cos_theta * ray.direction.z);

  CudaRay rotated_ray = cuda_make_ray(origin, direction, ray.time);

  if (!cuda_hittable_hit(*rotate.object, rotated_ray, t_values, record,
                         rand_state))
    return false;

  record.point = cuda_make_vec3(
      rotate.cos_theta * record.point.x + rotate.sin_theta * record.point.z,
      record.point.y,
      -rotate.sin_theta * record.point.x + rotate.cos_theta * record.point.z);

  record.normal = cuda_make_vec3(
      rotate.cos_theta * record.normal.x + rotate.sin_theta * record.normal.z,
      record.normal.y,
      -rotate.sin_theta * record.normal.x + rotate.cos_theta * record.normal.z);

  return true;
}

__device__ double cuda_rotate_y_pdf_value(const CudaRotateY &rotate,
                                          const CudaPoint3 &origin,
                                          const CudaVec3 &direction) {
  CudaPoint3 rotated_origin = cuda_make_vec3(
      rotate.cos_theta * origin.x - rotate.sin_theta * origin.z, origin.y,
      rotate.sin_theta * origin.x + rotate.cos_theta * origin.z);

  CudaVec3 rotated_direction = cuda_make_vec3(
      rotate.cos_theta * direction.x - rotate.sin_theta * direction.z,
      direction.y,
      rotate.sin_theta * direction.x + rotate.cos_theta * direction.z);

  return cuda_hittable_pdf_value(*rotate.object, rotated_origin,
                                 rotated_direction);
}

__device__ CudaVec3 cuda_rotate_y_random(const CudaRotateY &rotate,
                                         const CudaPoint3 &origin,
                                         curandState *state) {
  CudaPoint3 rotated_origin = cuda_make_vec3(
      rotate.cos_theta * origin.x - rotate.sin_theta * origin.z, origin.y,
      rotate.sin_theta * origin.x + rotate.cos_theta * origin.z);

  CudaVec3 obj_dir =
      cuda_hittable_random(*rotate.object, rotated_origin, state);

  return cuda_make_vec3(
      rotate.cos_theta * obj_dir.x + rotate.sin_theta * obj_dir.z, obj_dir.y,
      -rotate.sin_theta * obj_dir.x + rotate.cos_theta * obj_dir.z);
}

#endif // USE_CUDA