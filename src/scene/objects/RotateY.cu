
#include <hip/hip_runtime.h>
#ifdef USE_CUDA

#include "../../core/Hittable.cuh"
#include "RotateY.cuh"

__device__ bool cuda_rotate_y_hit(const CudaRotateY &rotate, const CudaRay &ray,
                                  CudaInterval t_values, CudaHitRecord &record,
                                  curandState *rand_state) {
  CudaPoint3 origin = cuda_make_vec3(
      rotate.cos_theta * ray.origin.x - rotate.sin_theta * ray.origin.z,
      ray.origin.y,
      rotate.sin_theta * ray.origin.x + rotate.cos_theta * ray.origin.z);

  CudaVec3 direction = cuda_make_vec3(
      rotate.cos_theta * ray.direction.x - rotate.sin_theta * ray.direction.z,
      ray.direction.y,
      rotate.sin_theta * ray.direction.x + rotate.cos_theta * ray.direction.z);

  CudaRay rotated_ray = cuda_make_ray(origin, direction, ray.time);

  if (!cuda_hittable_hit(*rotate.object, rotated_ray, t_values, record,
                         rand_state))
    return false;

  record.point = cuda_make_vec3(
      rotate.cos_theta * record.point.x + rotate.sin_theta * record.point.z,
      record.point.y,
      -rotate.sin_theta * record.point.x + rotate.cos_theta * record.point.z);

  record.normal = cuda_make_vec3(
      rotate.cos_theta * record.normal.x + rotate.sin_theta * record.normal.z,
      record.normal.y,
      -rotate.sin_theta * record.normal.x + rotate.cos_theta * record.normal.z);

  return true;
}

__device__ double cuda_rotate_y_pdf_value(const CudaRotateY &rotate,
                                          const CudaPoint3 &origin,
                                          const CudaVec3 &direction) {
  CudaPoint3 rotated_origin = cuda_make_vec3(
      rotate.cos_theta * origin.x - rotate.sin_theta * origin.z, origin.y,
      rotate.sin_theta * origin.x + rotate.cos_theta * origin.z);

  CudaVec3 rotated_direction = cuda_make_vec3(
      rotate.cos_theta * direction.x - rotate.sin_theta * direction.z,
      direction.y,
      rotate.sin_theta * direction.x + rotate.cos_theta * direction.z);

  return cuda_hittable_pdf_value(*rotate.object, rotated_origin,
                                 rotated_direction);
}

__device__ CudaVec3 cuda_rotate_y_random(const CudaRotateY &rotate,
                                         const CudaPoint3 &origin,
                                         curandState *state) {
  CudaPoint3 rotated_origin = cuda_make_vec3(
      rotate.cos_theta * origin.x - rotate.sin_theta * origin.z, origin.y,
      rotate.sin_theta * origin.x + rotate.cos_theta * origin.z);

  CudaVec3 obj_dir =
      cuda_hittable_random(*rotate.object, rotated_origin, state);

  return cuda_make_vec3(
      rotate.cos_theta * obj_dir.x + rotate.sin_theta * obj_dir.z, obj_dir.y,
      -rotate.sin_theta * obj_dir.x + rotate.cos_theta * obj_dir.z);
}

#endif // USE_CUDA