
#include <hip/hip_runtime.h>
#ifdef USE_CUDA

#include "Texture.cuh"

__device__ CudaColor CudaCheckerTexture::value(double u, double v,
                                               const CudaPoint3 &p) const {
  double inv_scale = 1.0 / scale;

  // Scale the point down to the texture's scale, in all three dimensions.
  int x_index = int(floor(inv_scale * p.x));
  int y_index = int(floor(inv_scale * p.y));
  int z_index = int(floor(inv_scale * p.z));

  // Use this to determine which checker space we're on.
  bool is_even = (x_index + y_index + z_index) % 2 == 0;

  return is_even ? even_texture->value(u, v, p) : odd_texture->value(u, v, p);
}

__device__ CudaColor CudaTexture::value(double u, double v,
                                        const CudaPoint3 &p) const {
  switch (type) {
  case CudaTextureType::TEXTURE_SOLID:
    return solid->value(u, v, p);

  case CudaTextureType::TEXTURE_NOISE:
    return noise->value(u, v, p);

  case CudaTextureType::TEXTURE_CHECKER:
    return checker->value(u, v, p);
  }
  // ERROR: Texture.cu::value - Unknown texture type in switch statement. This
  // should never happen in well-formed code.
  return CudaColor(0, 0, 0); // Safe fallback for GPU device code.
}

__device__ CudaTexture cuda_make_solid_texture(CudaColor albedo) {
  CudaTexture texture;
  texture.type = CudaTextureType::TEXTURE_SOLID;
  texture.solid = new CudaSolidColorTexture(albedo);
  return texture;
}

__device__ CudaTexture
cuda_make_checker_texture(double scale, const CudaTexture *even_texture,
                          const CudaTexture *odd_texture) {
  CudaTexture texture;
  texture.type = CudaTextureType::TEXTURE_CHECKER;
  texture.checker = new CudaCheckerTexture(scale, even_texture, odd_texture);
  return texture;
}

__device__ CudaTexture cuda_make_noise_texture(double scale,
                                               CudaPerlinNoise perlin) {
  CudaTexture texture;
  texture.type = CudaTextureType::TEXTURE_NOISE;
  texture.noise = new CudaNoiseTexture(scale, perlin);
  return texture;
}

#endif // USE_CUDA