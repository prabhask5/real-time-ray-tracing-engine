#include "hip/hip_runtime.h"
#include "Interval.cuh"

#ifdef USE_CUDA

// Batch interval operations for better GPU utilization.

__global__ void cuda_clamp_values_kernel(double *values,
                                         const CudaInterval *intervals,
                                         int count) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < count) {
    values[idx] = intervals[idx].clamp(values[idx]);
  }
}

void cuda_batch_clamp_values(double *d_values, const CudaInterval *d_intervals,
                             int count) {
  int blockSize = 256;
  int numBlocks = (count + blockSize - 1) / blockSize;
  cuda_clamp_values_kernel<<<numBlocks, blockSize>>>(d_values, d_intervals,
                                                     count);
  hipDeviceSynchronize();
}

// Optimized interval intersection.
__global__ void cuda_intersect_intervals_kernel(const CudaInterval *intervals1,
                                                const CudaInterval *intervals2,
                                                CudaInterval *result,
                                                int count) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < count) {
    double min_val = fmax(intervals1[idx].min, intervals2[idx].min);
    double max_val = fmin(intervals1[idx].max, intervals2[idx].max);
    result[idx] = CudaInterval(min_val, max_val);
  }
}

void cuda_batch_intersect_intervals(const CudaInterval *d_intervals1,
                                    const CudaInterval *d_intervals2,
                                    CudaInterval *d_result, int count) {
  int blockSize = 256;
  int numBlocks = (count + blockSize - 1) / blockSize;
  cuda_intersect_intervals_kernel<<<numBlocks, blockSize>>>(
      d_intervals1, d_intervals2, d_result, count);
  hipDeviceSynchronize();
}

#endif // USE_CUDA