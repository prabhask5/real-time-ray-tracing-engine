
#include <hip/hip_runtime.h>
#ifdef USE_CUDA

#include "../utils/math/Utility.cuh"
#include "../utils/memory/CudaMemoryUtility.cuh"
#include "Hittable.cuh"
#include "HittableList.cuh"
#include <assert.h>
#include <iomanip>
#include <iostream>
#include <sstream>

__device__ bool cuda_hittable_list_hit(const CudaHittableList &list,
                                       const CudaRay &ray, CudaInterval t_range,
                                       CudaHitRecord &out_rec,
                                       curandState *rand_state) {
  CudaHitRecord temp_rec;
  bool hit_anything = false;
  double closest_so_far = t_range.max;

  for (int i = 0; i < list.count; i++) {
    if (cuda_hittable_hit(list.hittables[i], ray,
                          cuda_make_interval(t_range.min, closest_so_far),
                          temp_rec, rand_state)) {
      hit_anything = true;
      closest_so_far = temp_rec.t;
      out_rec = temp_rec;
    }
  }

  return hit_anything;
}

__device__ double cuda_hittable_list_pdf_value(const CudaHittableList &list,
                                               const CudaPoint3 &origin,
                                               const CudaVec3 &direction) {
  // For a list of hittable objects, the pdf value biasing should be just the
  // average of all the PDF value biasing of the inner hittable objects.

  if (list.count == 0)
    return 0.0;

  double weight = 1.0 / list.count;
  double sum = 0.0;

  for (int i = 0; i < list.count; i++) {
    sum +=
        weight * cuda_hittable_pdf_value(list.hittables[i], origin, direction);
  }

  return sum;
}

__device__ CudaVec3 cuda_hittable_list_random(const CudaHittableList &list,
                                              const CudaPoint3 &origin,
                                              curandState *state) {
  // Randomly chooses one object in the list and returns a direction vector
  // sampled from it.

  int i = (int)(cuda_random_double(state) * list.count);
  i = (i < 0) ? 0 : ((i >= list.count) ? list.count - 1 : i);
  return cuda_hittable_random(list.hittables[i], origin, state);
}

// JSON serialization function for CudaHittableList.
std::string cuda_json_hittable_list(const CudaHittableList &obj) {
  std::ostringstream oss;
  oss << std::fixed << std::setprecision(6);
  oss << "{";
  oss << "\"type\":\"CudaHittableList\",";
  oss << "\"address\":\"" << &obj << "\",";
  oss << "\"hittables\":[";
  if (obj.hittables && obj.count > 0) {
    CudaHittable *host_hittables = new CudaHittable[obj.count];
    cudaMemcpyDeviceToHostSafe(host_hittables, obj.hittables, obj.count);
    for (int i = 0; i < obj.count; ++i) {
      if (i > 0)
        oss << ",";
      oss << cuda_json_hittable(host_hittables[i]);
    }
    delete[] host_hittables;
  }
  oss << "],";
  oss << "\"count\":" << obj.count << ",";
  oss << "\"bbox\":" << cuda_json_aabb(obj.bbox);
  oss << "}";
  return oss.str();
}

#endif // USE_CUDA